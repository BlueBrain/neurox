
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <stdio.h>

void print_gpu_memory_usage() {
    size_t free_byte;
    size_t total_byte;

    hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);

    if (hipSuccess != cuda_status) {
        printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status));
        exit(1);
    }

    double free_db = (double)free_byte;
    double total_db = (double)total_byte;
    double used_db = total_db - free_db;
    printf("\n  => GPU MEMORY USAGE (MB) : Used = %f, Free = %f MB, Total = %f",
           used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
    fflush(stdout);
}

void start_cuda_profile() {
    hipProfilerStart();
    print_gpu_memory_usage();
}

void stop_cuda_profile() {
    hipProfilerStop();
    print_gpu_memory_usage();
}
